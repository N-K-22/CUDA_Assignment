
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define _XOPEN_SOURCE 700
long long nsecs() {
    struct timespec t;
    clock_gettime(CLOCK_MONOTONIC, &t);
    return t.tv_sec*1000000000 + t.tv_nsec;
}

__global__ void matrixMultiplication ( int * A , int * B , int * C , int width ) {
    // TODO : Implement matrix multiplication kernel
    int component = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int sharedA[16][16];
    __shared__ int sharedB[16][16];
    //int sum = 0; //each thread does one component in the output matrix
   for (int i = 0; i < (width+15)/16; i++){
        
        if (row < width && (i*16 + threadIdx.x) < width){ // sets the component 

            sharedA[threadIdx.y][threadIdx.x] = A[row*width + i*16 + threadIdx.x]; //if the component falls within the area of shared memory covered by a thread

        } else{
            sharedA[threadIdx.y][threadIdx.x] = 0; // leave other parts blank
        }

        if ((i*16 + threadIdx.y) < width && col < width){
            sharedB[threadIdx.y][threadIdx.x] = B[(i*16 + threadIdx.y)*width + col]; //if matrix coordinate falls within the area of shared memory covered by a thread
        }
        else{
            sharedB[threadIdx.y][threadIdx.x] = 0;
        }
        __syncthreads();
        for (int  i = 0; i< 16; i++){
            component += sharedA[threadIdx.y][i] * sharedB[i][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < width && col < width){
    
            C[row*width + col] = component;
       

    }
   
}
void matrixMultiplicationCPUVersion(int* a, int* b, int* c, int width){
    int component;
    for(int row = 0; row < width; row++){
        for(int column = 0; column < width; column++){
            component = 0;
            for(int i = 0; i < width; i++){
                component += a[row*width + i] * b[i*width+column];
            }
            c[row*width +column] = component;
        }
    }

}
int main () {
    const int width = 128; // Matrix width
    int A [ width * width ] , B [ width * width ] , C [ width * width ], D [width*width]; // Host matrices
    int *d_A, *d_B, *d_C;

    // TODO : Initialize matrices ’A’ and ’B’ with random values
    for (int i =0; i < width*width; i++){
        A[i] = rand();
        B[i] = rand();
    }
    hipEvent_t GPU_start, GPU_stop;
    float GPU_time = 0;
    hipEventCreate(&GPU_start);
    hipEventCreate(&GPU_stop);
    hipEventRecord(GPU_start);

    // TODO : Allocate device memory for matrices ’A ’, ’B ’, and ’C ’
    //int *d_A;
   // int *d_B;
   // int *d_C;
   // int *d_D;
    
    hipMallocManaged((void **)&d_A, width*width*sizeof(int));
    hipMallocManaged((void**)&d_B, width*width*sizeof(int));
    hipMallocManaged((void**)&d_C, width*width*sizeof(int));
    //cudaMallocManaged((void**)&d_D, width*width*sizeof(int));

    // TODO : Copy matrices 'A' and 'B' from host to device

    hipMemcpy(d_A, A, width * width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B,B, width * width * sizeof(int), hipMemcpyHostToDevice);


    // Define grid and block dimensions
    dim3 dimGrid (( width + 15) / 16 , ( width + 15) / 16 , 1);
    dim3 dimBlock (16 , 16 , 1);

    // Launch the matrix multiplication kernel
    matrixMultiplication<<<dimGrid , dimBlock, 2*16*16*sizeof(int) >>>(d_A , d_B , d_C , width );

    // TODO : Copy the result matrix ’C ’ from device to host
    hipMemcpy(C, d_C, width*width*sizeof(int), hipMemcpyDeviceToHost);

    //timing the GPU kernel
    hipEventRecord(GPU_stop);
    hipEventSynchronize (GPU_stop); //waits for GPU_stop to complete
    hipEventElapsedTime(&GPU_time, GPU_start, GPU_stop);
    printf("Time taken for GPU version: %ld ms\n", GPU_time);



//CPU Matrix Multiplication
    long long start, end;
    long long cpu_time_used;
    start = nsecs();
    matrixMultiplicationCPUVersion(A, B, D, width);
    end = nsecs();
    cpu_time_used = end - start;
    printf("Time taken for CPU version: %ld ns\n", cpu_time_used);
    //memcpy(D, d_D, width*width*sizeof(int));

    // TODO : Verify the correctness of the result


    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess){
        printf("Error: %s\n", hipGetErrorString(error));
    }
    
    for (int i = 0; i < width*width; i++){
        if(D[i] != C[i]){
            printf("Value for CPU and GPU does not align at %d \n", i);
            printf("C: %d, D: %d \n", C[i], D[i]);

            break;
        }
    }
    

    // TODO : What is needed here ? --> Free memory

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    //free(d_D);
    return 0;

}
